
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

int main(void){
	hipDeviceProp_t prop; // initialise cudaDevice Info struct as object 'prop'
	int count;
	hipGetDeviceCount(&count);
	for(int i =0; i< count; i++){
	hipGetDeviceProperties(&prop, i);
	// now that the device properties are loaded into the struct
			// let's see what we can find

	// some things that might be interesting
	cout<<"Name of GPU: "<<prop.name<<endl;
	
	}//end of for loop

}
