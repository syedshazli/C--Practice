#include <iostream>
// find more on page 30 of the book

using namespace std;

int main(void){
	hipDeviceProp_t prop; // initialise cudaDevice Info struct as object 'prop'
	int count;
	hipGetDeviceCount(&count);
	for(int i =0; i< count; i++){
	hipGetDeviceProperties(&prop, i);
	// now that the device properties are loaded into the struct
			// let's see what we can find

	// some things that might be interesting
	cout<<"---General informaiton for device---"<<endl;
	cout<<"Name of GPU: "<<prop.name<<endl;
	cout<<"Compute capability: "<<prop.major<<"and " <<prop.minor<<endl;	    cout<<"Clock rate: "<<prop.clockRate<<endl;
	
	cout<<"---Memory Information---"<<endl;
	cout<<"Total global memory: "<<prop.totalGlobalMem<<" bytes"<<endl;
	cout<<"Total constant memory: "<<prop.totalConstMem<<endl;
	cout<<"Max mem pitch: "<<prop.memPitch<<endl;
	
	cout<<"---Multiprocessor Info for device "<<i<<endl;
	cout<<"Multiprocessor count: "<<prop.multiProcessorCount<<endl;
	cout<<"Shared memory per multiprocessor: "<<prop.sharedMemPerBlock<<" bytes"<<endl;
	cout<<"32 bit Registers per multiprocessor: "<<prop.regsPerBlock<<endl;
	cout<<"Threads in warp: "<<prop.warpSize<<endl;
	cout<<"Max threads per block: "<<prop.maxThreadsPerBlock<<endl;

	}//end of for loop

}
