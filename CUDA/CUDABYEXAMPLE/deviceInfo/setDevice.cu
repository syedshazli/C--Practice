
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

int main(void){
hipDeviceProp_t prop;
int dev;

hipGetDevice(&dev);
cout<<"ID of current CUDA Device: "<< dev<<endl;;

memset(&prop, 0, sizeof(hipDeviceProp_t) );
prop.major = 1;
prop.minor = 3;
hipChooseDevice(&dev, &prop);
cout<<"ID of CUDA Device closest to revision 1.3: "<<dev;
hipSetDevice(dev);

}
