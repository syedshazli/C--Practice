
#include <hip/hip_runtime.h>
#define N 10 // 10 threads

// note that all of this code is run on the CPU, no device code written
// pass by reference
void add(int *a, int *b, int *c){

	// takes up a lot more code in the CPU to make sure this runs in paralell
	// especially if we wanted one core to work on say, odd numbered
	int tid = 0;
	while (tid < N){
	c[tid] = a[tid] + b[tid];
        tid +=1; // increment by one because of one CPU	
	
	
	}

}




int main(void){

	int a[N], b[N], c[N];


	// fill arrays on the CPU
	for (int i = 0; i<N; i++){
	
		a[i] = -i;
		b[i] = i * i;
	}
	
	add(a,b,c);
	
	for(int i = 0; i<N; i++){
		
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}



	return 0;
}


