#define N 10


// now run the add function in paralell on the GPU
int main(void){

	
	int a[N], b[N], c[N];
	int *dev_a, int *dev_b, int *dev_c; // pointers to device code

	hipMalloc( (void **)&dev_a, N*sizeof(int) );
	hipMalloc( (void **)&dev_b, N*sizeof(int) );
	hipMalloc( (void **)&dev_c, N*sizeof(int) );


	// fill arrays a and b on the CPU
	for(int i = 0; i<N; i++){
		a[i] = -i;
		b[i] = i*i;
	}


	// copy arrays a and b to the GPU using hipMemcpy
	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

	add<<<N,1>>>(dev_a, dev_b, dev_c);//<<<N,1>>>, N specifies thread block
	// and we're using 1 block

	// copy arrat 'c' back from GPU to CPU
	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);



	// display the result
	for(int i = 0; i<N; i++){
	
		printf("%d + %d = %d\n", a[i], b[i], c[i] );
	
	}// end of for loop
	
	// free allocated memory
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	return 0 ;
}

