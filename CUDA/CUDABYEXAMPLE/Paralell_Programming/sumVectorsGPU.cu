
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10


__global__ void add(int *dev_a, int *dev_b, int *dev_c){
	
	// no need to define blockIdx, built in that CUDA runtime defines
	// contains value for block index for whatever block is executing
	// so block 5 is executing, blockIdx.x returns 5, which is the TID
	// when initialising the block in the host, N is 0 indexed
	// so tid goes from 0 to N-1
	// as a result, each block computes 1 value in the dev_c array
	// if we have more entries in the array than threads, we specify by
	// for(int i = tid, i <tid+2, i++){if(tid<N{dev_c[tid] = ....}}
	
	int tid = blockIdx.x; // handle data @ this index
	if(tid < N){
	
		dev_c[tid] = dev_a[tid] + dev_b[tid];
	}
}


// now run the add function in paralell on the GPU
int main(void){

	
	int a[N];
        int b[N];
        int c[N];
	int *dev_a;
       	int *dev_b; 
	int *dev_c; // pointers to device code

	hipMalloc( (void **)&dev_a, N*sizeof(int) );
	hipMalloc( (void **)&dev_b, N*sizeof(int) );
	hipMalloc( (void **)&dev_c, N*sizeof(int) );


	// fill arrays a and b on the CPU
	for(int i = 0; i<N; i++){
		a[i] = -i;
		b[i] = i*i;
	}


	// copy arrays a and b to the GPU using cudaMemcpy
	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

	// note how device arrays haven't been modified till we call device function
	
	// similar to making 10 copies of the add function and executing in paralell
	// each paralell invocation is a block
	add<<<N,1>>>(dev_a, dev_b, dev_c);//<<<N,1>>>, N specifies number of paralell thread blocks
	
	// collection of paralell blocks: a grid
	// in our case, we have a 1D grid of N blocks


	// and we're using 1 block

	// copy arrat 'c' back from GPU to CPU
	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);



	// display the result
	for(int i = 0; i<N; i++){
	
		printf("%d + %d = %d\n", a[i], b[i], c[i] );
	
	}// end of for loop
	
	// free allocated memory
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	return 0 ;
}

