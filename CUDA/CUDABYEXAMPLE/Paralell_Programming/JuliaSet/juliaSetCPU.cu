


/**
  * The Julia Set evaluates a iterative equation for points in a complex plane
  *A point isn't in the set if iterating the equation diverges for that point
  * AKA grows towards infinity
  * Otherwise, it's in the set
  */


// a single precision component r
// a single precision imaginary component i
// defines addition and muktiplication operators
struct hipComplex{

	float r;
	float i;

	// initialise constructor
	hipComplex(float a, float b) : r(a), i(b){}
	
	float magnitude2(void) {return r * r + i * i;}

	hipComplex operator*(const hipComplex& a){
	
		return hipComplex (r*a.r - i*a.i, i*a.r + r*a.i);
	}


	hipComplex operator+(const hipComplex& a){
	
	return hipComplex(r+a.r, i+a.i);	
	}


};


// translate pixel coordinate to coordinate in complex space
// shift by DIM/2 to center the complex plane to the center
// want image ti span in -1,  1, so we scale by DIM/2
// which explains the jx, jy
// if we want to zoom in or out, we adjust the scale factor
// we find out if the number in a julia set by computing the equation in the book
// can adjust the constant c from the hipComplex object


// 200 iterations, does the result exceed a threshold?
// if so, return 0, otherwise, if all iterations finished, return 1;
int julia(int x, int y){

	const float scale  = 1.5;
	float jx  = scale * (float) (DIM/2 - x)/ (DIM/2);
	float jy = scale * (float) (DIM/2 -y) / (DIM/2);

	hipComplex c(-0.8, 0.156);
	hipComplex a (jx, jy);

	int i = 0;
	for(i = 0; i<200; i++){
	
		a = a*a +c;
		if(a.magnitude2()>1000)
		{return 0;}
	}
	
	return 1;
}



void kernel(unsigned char *ptr){

	// iterate through all points we want to render
	// calling julia on each to determine if it's in the juliaSet
	// julia returns 1 if the point is in the set and 0 if not
	// point color is red if julia is 1 and black otherwise
	for(int y = 0; y<DIM; y++){
	
	   for(int x  = 0; x<DIM; x++){
	   
	   
		   int offset = x+y * DIM;
		   int juliaValue = julia(x,y);

		  ptr[offset*4+0] = 255 * juliaValue;
		  ptr[offset*4 +1] = 0;
		  ptr[offset*4 +2] = 0;
		  ptr[offset*4 +3] = 255;


	   }
	}


}


int main(void){

	CPUBitmap bitmap(DIM, DIM);
	unsigned char *ptr = bitmap.get_ptr();

	kernel(ptr);

	bitmap.display_and_exit();

}
