#include "hip/hip_runtime.h"

#define DIM 1000

struct hipComplex {
 float r;
 float i;
 hipComplex( float a, float b ) : r(a), i(b) {}
 __device__ float magnitude2( void ) {
 return r * r + i * i;
 }
 __device__ hipComplex operator*(const hipComplex& a) {
 return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
 }
 __device__ hipComplex operator+(const hipComplex& a) {
 return hipComplex(r+a.r, i+a.i);
 }
};


// device means the code will run on the GPU, not the host
// only callable from other device function or from global functions
// so our main cannot call a function from device
__device__ int julia( int x, int y ) {
 const float scale = 1.5;
 float jx = scale * (float)(DIM/2 - x)/(DIM/2);
 float jy = scale * (float)(DIM/2 - y)/(DIM/2);
 hipComplex c(-0.8, 0.156);
 hipComplex a(jx, jy);
 int i = 0;
 for (i=0; i<200; i++) {
 a = a * a + c;
 if (a.magnitude2() > 1000)
 return 0;
 }
 return 1;
}


// runs on GPU, exact same CPU implementation
__global__ void kernel(unsigned char *ptr){

        // iterate through all points we want to render
        // calling julia on each to determine if it's in the juliaSet
        // julia returns 1 if the point is in the set and 0 if not
// map from threadIdx/BlockIdx to pixel position
 int x = blockIdx.x;
 int y = blockIdx.y;
 int offset = x + y * gridDim.x;
 // now calculate the value at that position
 int juliaValue = julia( x, y );
 ptr[offset*4 + 0] = 255 * juliaValue;
 ptr[offset*4 + 1] = 0;
 ptr[offset*4 + 2] = 0;
 ptr[offset*4 + 3] = 255;

}



int main(void){

CPUBitmap bitmap( DIM, DIM );
 
// hold a copy of data on device
unsigned char *dev_bitmap;
  
 // allows us to hold data
 hipMalloc( (void**)&dev_bitmap,
 		bitmap.image_size() );


 // allows us to compute over 2D domain
 // we're in 2d, so 3rd dimension gas vakye if 1
 dim3 grid(DIM,DIM);

 // each point can be compared independently (SIMD)
 // one copy of functions for each point we wanna compute
 kernel<<<grid,1>>>( dev_bitmap );
 
 // allows us to copy results back to host
 hipMemcpy( bitmap.get_ptr(),
 		dev_bitmap,
 		bitmap.image_size(),
 		hipMemcpyDeviceToHost  );
 
 bitmap.display_and_exit();
 hipFree( dev_bitmap ); 

}
