
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


// host must do cuda malloc on arrays a,b, c
// then a CUDAMEMCPY on array c

using namespace std;

__global__ void matmul(int a[4][4], int b[4][4], int c[4][4], int N){

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	
	if(row<N && col < N)
	{
	int val = 0;
	for(int k = 0; k < N; k++){
		
		val += a[row][k] * b[k][col];	
	
	}
	
	c[row][col] = val;
	}

}

int main(){
	int a[4][4] = {
        1, 1, 1, 1,
        1, 1, 1, 1,
        1, 1, 1, 1,
        1, 1, 1, 1,
    };

	 int b[4][4] = {
        2, 2, 2, 2,
        2, 2, 2, 2,
        2, 2, 2, 2,
        2, 2, 2, 2,
    };

	int c[4][4] = {
	0, 0, 0, 0,
	0, 0, 0, 0,
	0, 0, 0, 0,
	};

	int (*dev_c)[4];
	dev_c = c;

	// allocate with CUDA MALLOC
	hipMalloc( (void**) &dev_c, sizeof(c));

	// 4 blocks, 4 threads per block
	matmul<<<4,4>>> (a,b,dev_c, 4);

	// finished computation, store result in dev_c
	int host_c[4][4];
	hipMemcpy(&host_c, dev_c, sizeof(c), hipMemcpyDeviceToHost);


	for(int row  = 0; row <4; row++ ){
		
     	   for(int col = 0; col<4; col++){//c++ XD

        	cout<<host_c[row][col]<<','<<' ';

        }
        cout<<endl;
    }
	
}
