
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


// host must do cuda malloc on arrays a,b, c
// then a CUDAMEMCPY on array c

using namespace std;

__global__ void matmul(int a[4][4], int b[4][4], int c[4][4], int N){

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	
	if(row<N && col < N)
	{
	int val = 0;
	for(int k = 0; k < N; k++){
		
		val += a[row][k] * b[k][col];	
	
	}
	
	c[row][col] = val;
	}

}

int main(){
	int a[4][4] = {
        1, 1, 1, 1,
        1, 1, 1, 1,
        1, 1, 1, 1,
        1, 1, 1, 1,
    };

	 int b[4][4] = {
        2, 2, 2, 2,
        2, 2, 2, 2,
        2, 2, 2, 2,
        2, 2, 2, 2,
    };

	int c[4][4] = {
	0, 0, 0, 0,
	0, 0, 0, 0,
	0, 0, 0, 0,
	};

	int (*dev_c)[4];// points to the first row of the array, each row has 4 ints
//	dev_c = c; // dev_c points to the 2d array c

	// allocate with CUDA MALLOC
	hipMalloc( (void**) &dev_c, sizeof(c));
	
	int(*dev_a)[4];
	//dev_a = a;
	
	int(*dev_b)[4];
	//dev_b = b;

	hipMalloc( (void**) &dev_a, sizeof(a)  );
	hipMalloc((void**) &dev_b, sizeof(b) );
	
//	dev_a = a;
//	dev_b = b;

	hipMemcpy(dev_b,b,sizeof(b),hipMemcpyHostToDevice);
	hipMemcpy(dev_a,a,sizeof(a),hipMemcpyHostToDevice);
	
	// 4 blocks, 4 threads per block
	matmul<<<dim3(2,2),dim3(2,2)>>> (dev_a,dev_b,dev_c, 4);

	// finished computation, store result in dev_c
	int host_c[4][4];
	hipMemcpy(host_c, dev_c, sizeof(c), hipMemcpyDeviceToHost);


	for(int row  = 0; row <4; row++ ){
		
     	   for(int col = 0; col<4; col++){//c++ XD

        	cout<<host_c[row][col]<<','<<' ';

        }
        cout<<endl;
    }
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);	
}
