
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

// can pass parameters to kernel
// allocate memory when doing anything with device

// device code
// run on the GPU


// ctrl + shift + v to copy paste

__global__ void add(int a, int b, int *c){
	*c = a + b; // takes memory that c is pointing to
			// stores result of a + bi

	// in the dvice: can pass host pointers around, but cant use host pointer to access memory from host code
	// and device pointers can access memory from device code

}

int main(void){

	int c;
	int *dev_c;

	//CUDA Malloc overview
	// first argument: point to a pointer you want to hold address of
	//					  newly allocated memory

	// second parameter: size of allocation needed to make
	// cannot use pointer returned by CUDA malloc to read/write from mem
	//						in the host

	/**
	  * Allowed: Passing pointers allocated w/cudaMalloc() to functions that execute on device
	  * ALlowed: use pointers allocated w/cudaMalloc to read/write memory from code that executes on device
	  * NOT ALLOWED: Use pointers allocated w.cudaMalloc to read/write memory from code that executes on host
	  * CANNOT use free() to release mempry w/cuda malloc
	  * Instead, just use cudaFree()
	  */

	hipMalloc( (void**)&dev_c, sizeof(int));//behaves similarly to malloc()
	//tells CUDA runtime to allocated memory on the device (GPU)




	add<<<1,1>>>(2, 7, dev_c);// passing parameters using () to kernel, just have <<1,1>>

	// reminder when passing in pointer without the *, it is just a memory address
			
	
	// access memort on the device via calls to cudaMemcpy
	// like C memcpy, one more parameter to specify source and destination pointers point to device memory
	// in this example, we tell runtime the source pointer is device pointer, destination pointer is host pointer
	// can use cudaMemcpyHosttoDeivce for opposite effecct
	// tell them that both poiunters are on the device by saying cudaMemcpyDevicetoDevice
	// if both on the host, just use memcpy() from C
			hipMemcpy(&c,
				dev_c,
				sizeof(int),
				hipMemcpyDeviceToHost);
	printf("2 + 7 = %d\n", c);
	hipFree(dev_c);

	int count;
	hipGetDeviceCount(&count); // get count of cuda devices
	
	return 0;


}


