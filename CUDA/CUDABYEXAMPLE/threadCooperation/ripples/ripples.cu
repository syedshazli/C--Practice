#include "hip/hip_runtime.h"


__global__ void kernel(unsigned char *ptr, int ticks){

	
	// map from threadIDx/BlockIDx to pixel pos
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;


	// now calculate the value at that position
	float fx = x-DIM/2;
	float fy = y - DIM/2;
	float d = sqrtf(fx*fx+ fy*fy);





}










