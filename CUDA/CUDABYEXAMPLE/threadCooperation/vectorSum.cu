
#include <hip/hip_runtime.h>
#include <iostream>
#define N 20
using namespace std;

__global__ void add(int *a, int *b, int *c){

	int threadID  = threadIdx.x;
	if(threadID < N){
	
		c[threadID] = a[threadID] + b[threadID];
	
	}

	}


int main(void){
	

	int a[N], b[N], c[N];
	int *device_a, *device_b, *device_c;

	//allocate memory on the device
	hipMalloc( (void**)&device_a, N*sizeof(int)  );
	hipMalloc( (void**)&device_b, N* sizeof(int)  );
	hipMalloc( (void**)&device_c, N*sizeof(int) );

	for(int i = 0; i<N; i++){
	
		a[i] = i;
		b[i] = i*i;
	}


	// copy arrays a and b to the GPU using cudamemcpy

	hipMemcpy(device_a, a, N*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy(device_b, b, N*sizeof(int), hipMemcpyHostToDevice);

	// compute the addition
	// 1 thread block, 20 threads on the block
	add<<<1,N>>>(device_a, device_b, device_c);

	// copy array c from GPU to CPu so we can view results
	hipMemcpy(c, device_c, N*sizeof(int), hipMemcpyDeviceToHost);

	for(int i = 0; i<N; i++){
	
		cout<<a[i]<< " + "<<b[i] <<" = "<<c[i]<<endl;
	}



	hipFree(device_a);
	hipFree(device_b);
	hipFree(device_c);
	return 0;
	}


