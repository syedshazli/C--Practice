#include "hip/hip_runtime.h"
#include <iostream>
#define N (20*487)
using namespace std;

__global__ void add(int *a, int *b, int *c, int *d){

	int threadID  = threadIdx.x + blockIdx.x * blockDim.x;
	while(threadID < N){
	
		c[threadID] = (a[threadID] + b[threadID]) * d[threadID];
		threadID += blockDim.x * gridDim.x;
	}

	}


int main(void){

	hipDeviceProp_t prop;
	int whichDevice;
 	hipGetDevice( &whichDevice  );
 	hipGetDeviceProperties( &prop, whichDevice ) ;
	if(!prop.deviceOverlap){
	printf("No device overlap, sorry!");
	}	

	else{
	printf("Device overlap all set");
	}

	int a[N], b[N], c[N], d[N];
	int *device_a, *device_b, *device_c, *device_d;
	//allocate memory on the device
	hipMalloc( (void**)&device_a, N*sizeof(int)  );
	hipMalloc( (void**)&device_b, N* sizeof(int)  );
	hipMalloc( (void**)&device_c, N*sizeof(int) );
	hipMalloc( (void**)&device_d, N*sizeof(int) );

	for(int i = 0; i<N; i++){
	
		a[i] = i;
		b[i] = i*i;
		d[i] = (int)i/2;
	}


	// copy arrays a and b to the GPU using cudamemcpy

	hipMemcpy(device_a, a, N*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy(device_b, b, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_d, d, N*sizeof(int), hipMemcpyHostToDevice );
	

	// compute the addition
	// 1 thread block, 20 threads on the block
	add<<<128,128>>>(device_a, device_b, device_c, device_d);

	// copy array c from GPU to CPu so we can view results
	hipMemcpy(c, device_c, N*sizeof(int), hipMemcpyDeviceToHost);

	bool success = true;
	for(int i = 0; i<N; i++){
		
		if ( (a[i] + b[i] ) * d[i] !=c[i]){
		cout<<"ERROR!!!"<<endl;
		success = false;
		}

		//cout<<a[i]<< " + "<<b[i] <<" * "<<d[i]<<" = "<<c[i]<<endl;
	}
	
	if(success){
	cout<<"We did itttt"<<endl;
	}
	
	// print last 10 elements
	 for(int i = N-20; i<N; i++){


               cout<<a[i]<< " + "<<b[i] <<" * "<<d[i]<<" = "<<c[i]<<endl;        }


	hipFree(device_a);
	hipFree(device_b);
	hipFree(device_d);
	hipFree(device_c);
	
	return 0;
	}


