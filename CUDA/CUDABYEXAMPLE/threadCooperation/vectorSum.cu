
#include <hip/hip_runtime.h>
#include <iostream>
#define N 20
using namespace std;

__global__ void add(int *a, int *b, int *c, int *d){

	int threadID  = threadIdx.x;
	if(threadID < N){
	
		c[threadID] = (a[threadID] + b[threadID]) * d[threadID];
	
	}

	}


int main(void){
	

	int a[N], b[N], c[N], d[N];
	int *device_a, *device_b, *device_c, *device_d;
	//allocate memory on the device
	hipMalloc( (void**)&device_a, N*sizeof(int)  );
	hipMalloc( (void**)&device_b, N* sizeof(int)  );
	hipMalloc( (void**)&device_c, N*sizeof(int) );
	hipMalloc( (void**)&device_d, N*sizeof(int) );

	for(int i = 0; i<N; i++){
	
		a[i] = i;
		b[i] = i*i;
		d[i] = (int)i/2;
	}


	// copy arrays a and b to the GPU using cudamemcpy

	hipMemcpy(device_a, a, N*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy(device_b, b, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_d, d, N*sizeof(int), hipMemcpyHostToDevice );
	

	// compute the addition
	// 1 thread block, 20 threads on the block
	add<<<1,N>>>(device_a, device_b, device_c, device_d);

	// copy array c from GPU to CPu so we can view results
	hipMemcpy(c, device_c, N*sizeof(int), hipMemcpyDeviceToHost);

	for(int i = 0; i<N; i++){
	
		cout<<a[i]<< " + "<<b[i] <<" * "<<d[i]<<" = "<<c[i]<<endl;
	}



	hipFree(device_a);
	hipFree(device_b);
	hipFree(device_d);
	hipFree(device_c);
	
	return 0;
	}


