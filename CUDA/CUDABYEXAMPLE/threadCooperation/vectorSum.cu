
#include <hip/hip_runtime.h>

#define N 20


__global__ void add(int *a, int *b, int *c){

	int threadID  = threadIdx.x;
	if(threadID < N){
	
		c[threadID] = a[threadID] + b[threadID];
	
	}

	}


int main(void){
	

	int a[N], b[N], c[N];
	int *device_a, *device_b, *device_c;

	//allocate memory on the device
	hipMalloc( (void**)&device_a, N*sizeof(int)  );
	hipMalloc( (void**)&device_b, N & sizeof(int)  );
	hipMalloc( (void**)&device_c, N*sizeof(int) );

	for(int i = 0; i<N; i++){
	
		a[i] = i;
		b[i] = i*i;
	}


	// copy arrays a and b to the GPU


	}


