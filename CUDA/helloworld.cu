
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

// using global void, we know function should run on a device 
// kernel function is given to compiler that handles device code, and feeds it to main
__global__ void kernel (void){
	
}

// to run: nvcc -o executable helloworld.cu
// this code runs entirely on the host
//	The CPU and system's memory is the host
//	the GPU and its memory is the device


// A function that executes on the device is called a kernel 
//(aka using a GPU to execute code)

int main(void){
	
	kernel<<<1,1>>>(); // <<<>>> arguments we pass to runtime system
	printf("Hello World!\n");
	return 0;
}
