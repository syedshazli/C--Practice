#include "hip/hip_runtime.h"

// a basic example of an exception being called in cuda-gdb
__global__ callMe(int *dev_a){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // exception thrown here.
    dev_a[tid+10000] = tid;
}


int main(){

    int *dev_a;
    int host_a[4];
    hipMalloc( (void**) &dev_a, sizeof(int)*4);
  
    callMe<<<1, 16>>> (dev_a);

    hipMemcpy(host_a, dev_a, 4*sizeof(int), hipMemcpyDeviceToHost);
}